#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void kernel(float*da,float*db,float*dc,int mw,int w){
    int i=blockIdx.x*blockDim.x+threadIdx.x;

    int s=i-(mw)/2;  // Determine the starting index for the current element
    float pv=0;  // Initialize the result for the current element
    
    // Convolution operation: sum of element-wise product between `da` and `db`
    for(int j=0;j<w;j++){
        if(s+j>=0&&s+j<w){  // Check if the indices are within bounds
            pv+=da[s+j]*db[j];  // Sum the products
        }
    }
    dc[i]=pv;  // Store the result in the output vector
}

int main(){
    int n1,n2;

    // Ask for the length of the input vector and mask
    printf("Length of the vector : ");
    scanf("%d",&n1);
    printf("Enter the length of mask : ");
    scanf("%d",&n2);

    // Allocate memory for the input vectors and the output vector
    float a[n1],b[n2],c[n1];
    float *da,*db,*dc;

    hipMalloc((void **)&da,n1*sizeof(float));
    hipMalloc((void **)&db,n2*sizeof(float));
    hipMalloc((void **)&dc,n1*sizeof(float));

    // Input the vectors a and b
    printf("Enter vector one : ");
    for(int i=0;i<n1;i++)
        scanf("%f",&a[i]);
    printf("Enter vector two (aka mask) : ");
    for(int i=0;i<n2;i++)
        scanf("%f",&b[i]);
    
    // Copy the input vectors to device memory
    hipMemcpy(da,a,n1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n2*sizeof(float),hipMemcpyHostToDevice);

    // Define the grid and block dimensions for CUDA
    dim3 grid(n1,1,1);
    dim3 blk(1,1,1);

    // Call the kernel to perform the convolution
    kernel<<<grid,blk>>>(da,db,dc,n2,n1);

    // Copy the result back to the host
    hipMemcpy(c,dc,n1*sizeof(float),hipMemcpyDeviceToHost);

    // Print the output vector
    for(int i=0;i<n1;i++)
        printf("%f\t",c[i]);
    printf("\n");

    // Free device memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

/*Length of the vector : 5
//Enter the length of mask : 3
Enter vector one : 1 2 3 4 5
Enter vector two (aka mask) : 0.2 0.5 0.2

0.400000   1.000000   1.600000   2.000000   1.600000
*/

