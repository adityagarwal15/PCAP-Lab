#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024  // Size of the vectors (can be modified)
#define THREADS_PER_BLOCK 256  // Fixed number of threads per block

// CUDA Kernel for vector addition
__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread index
    if (i < n)  // Prevent out-of-bounds access
        C[i] = A[i] + B[i];
}

int main() {
    int size = N * sizeof(float);

    // Allocate memory on host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = (N - i) * 1.0f;
    }

    // Allocate memory on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // **Calculate number of blocks**
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch kernel
    vectorAdd<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the first 10 results for verification
    printf("Result (First 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%.2f ", h_C[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

